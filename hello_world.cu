#include "hip/hip_runtime.h"
#include <iris/iris.hpp>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <stdio.h>

char a[12] = "hello world";
char b[12];
size_t size = 12;

extern "C" __global__ void uppercase(char* b, char* a) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  if (a[i] >= 'a' && a[i] <= 'z') b[i] = a[i] + 'A' - 'a';
  else b[i] = a[i];
}

int main(int argc, char** argv) {
  iris::Platform platform;
  platform.init(&argc, &argv, true);

  iris::Mem mem_a(size);
  iris::Mem mem_b(size);

  iris::Task task;
  task.h2d(&mem_a, 0, size, a);
  void* params[2] = { &mem_b, &mem_a };
  int params_info[2] = { iris_w, iris_r };
  task.kernel("uppercase", 1, NULL, &size, NULL, 2, params, params_info);
  task.d2h(&mem_b, 0, size, b);
  task.submit(iris_roundrobin, NULL, true);

  printf("%s\n", b);

  platform.finalize();

  return 0;
}